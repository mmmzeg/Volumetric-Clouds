#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

__host__ __device__ float generateRandomFloatFromSeed(int index, float seed){
  thrust::default_random_engine rng(hash(index*seed));
  thrust::uniform_real_distribution<float> u01(0,1);

  return (float)u01(rng);
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){

  //establish "right" camera direction
  glm::normalize(eye); glm::normalize(view);
  glm::vec3 right = glm::normalize(glm::cross(up, view));
  
  // calculate P1 and P2 in both x and y directions
  glm::vec3 image_center = eye + view;
  glm::vec3 P1_X = image_center - tan((float)4.0*fov.x)*right;
  glm::vec3 P2_X = image_center + tan((float)4.0*fov.x)*right;
  glm::vec3 P1_Y = image_center - tan((float)4.0*fov.y)*up;
  glm::vec3 P2_Y = image_center + tan((float)4.0*fov.y)*up;
  
  glm::vec3 bottom_left  = P1_X + (P1_Y - image_center);
  glm::vec3 bottom_right = P2_X + (P1_Y - image_center);
  glm::vec3 top_left     = P1_X + (P2_Y - image_center);

  glm::vec3 imgRight = bottom_right - bottom_left;
  glm::vec3 imgUp    = top_left - bottom_left;

  // supersample the pixels by taking a randomly offset ray in each iteration
  glm::vec3 random_offset = generateRandomNumberFromThread(resolution, time, x, y);
  float x_offset = random_offset.x;
  float y_offset = random_offset.y;
  glm::vec3 img_point = bottom_left + ((float)x + x_offset)/(float)resolution.x*imgRight + ((float)y + y_offset)/(float)resolution.y*imgUp;
  glm::vec3 direction = glm::normalize(img_point - eye); 

  // return value
  ray r; r.origin = eye; r.direction = direction;
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
  
	if(x<=resolution.x && y<=resolution.y){

		glm::vec3 color;
		color.x = image[index].x*255.0;
		color.y = image[index].y*255.0;
		color.z = image[index].z*255.0;

		if(color.x>255)
			color.x = 255;
		else if (color.x<0)
			color.x = 0;
		
		if(color.y>255)
			color.y = 255;
		else if (color.y<0)
			color.y = 0;
		
		if(color.z>255)
			color.z = 255;
		else if (color.z<0)
			color.z = 0;
      
		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 1.0;
		PBOpos[index].x = color.x;
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void voxelizeVolumeWithNoise(int index, volume* volumes, Perlin* perlin1, Perlin* perlin2, int timestep)
{
	// identify current volume
	volume V = volumes[index];

	// get index of voxel within volume
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int z = (blockIdx.z * blockDim.z) + threadIdx.z;
	int voxelIndex = x*V.xyzc.y*V.xyzc.z + y*V.xyzc.z + z;
	
	// find distance from voxel to center of volume
	glm::vec3 localPosition3D = getLocalVoxelPosition(glm::vec3((float)x, (float)y, (float)z), V);
	float length = glm::distance(localPosition3D, glm::vec3(0.0f, 0.0f, 0.0f));
	
	// get random number
	//float modifier = generateRandomFloatFromSeed(voxelIndex, perlin1->mSeed) * 0.5 - 0.25;
	float modifier = 0.0f;

	if ((length < 0.5f + modifier) && (voxelIndex < V.xyzc.x*V.xyzc.y*V.xyzc.z) && (localPosition3D.y < 0.2)) {
		float p1 = (perlin1->Get(multiplyMV(V.transform, glm::vec4(localPosition3D, 1.0))) + (1.0 - (length / (0.5f + modifier)))) * ((0.5f + modifier) - length);
		float p2 = (perlin2->Get(multiplyMV(V.transform, glm::vec4(localPosition3D, 1.0))) + (1.0 - (length / (0.5f + modifier)))) * ((0.5f + modifier) - length);
		V.voxels[voxelIndex].density = max(glm::mix(p1, p2, (float)(timestep%20) / 20.0f), 0.0f);
	}
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, glm::vec3* colors, light* lights, int numberOfLights,
							material* materials, volume* volumes, int numberOfVolumes, float iterations)
{
	// Find index of pixel and create empty color vector
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	// Get initial ray from camera through this position
	ray currentRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
  
	// Return values for the intersection test
	glm::vec3 intersection_point;

	// constant attenuation for transmission
	float k = 0.2f;

	// initialize color along ray to black
	colors[index] = cam.brgb;
	
	for (int v = 0; v < numberOfVolumes; v++)
	{
		// initialize transmission of pixel to 1.0
		float T = 1.0;
	
		glm::vec3 newColor = glm::vec3(0.0f);
		volume V = volumes[v];

		if (volumeIntersectionTest(V, currentRay, intersection_point) > 0.0)
		{
			// initial intersection point on bounding box of volume
			glm::vec3 marchPoint = intersection_point;

			// color of volumetric material 
			glm::vec3 volCol = materials[V.materialid].color;

			// index of initial intersection point in volume density grid
			int voxelIndex = getVoxelIndex(marchPoint, V);

			// recurse through the volume and perform operations
			// while still inside (i.e. point has valid voxel index)
			while (voxelIndex >= 0) {

				// density of voxel at point
				float p = V.voxels[voxelIndex].density;
			
				// transmission value at point evaluated using given function
				float deltaT = exp(-k*V.step*p);
			
				// accumulate transmission along ray
				// and break if below threshold
				T *= min(deltaT, 1.0f);
				if (T < 0.1) break;

				// calculate lighting
				if (deltaT < 1.0f) {
					for (int i = 0; i < numberOfLights; i++)
					{
						// initialize transmission along
						// light ray to zero
						float Q = 1.0;

						// ith scene light
						light L = lights[i];

						// material color scaled by light intensity
						glm::vec3 CF = volCol * L.color;

						// first sampling point along light ray is
						// march point
						glm::vec3 lightPoint = marchPoint;

						// light ray
						glm::vec3 lightDir = glm::normalize(L.position - marchPoint);
				
						// get index of voxel for point along light ray
						int lightVoxelIndex = getVoxelIndex(lightPoint, V);
				
						// recurse along light ray and perform operations
						// while still inside (i.e. point has valid voxel index
						while (lightVoxelIndex >= 0) 
						{
							// density at point along light ray
							float pLight = V.voxels[lightVoxelIndex].density;
					
							// light transmission value at point along light ray
							float deltaQ = exp(-k*V.step*pLight);

							// accumulate opacity of point
							Q *= deltaQ;
							if (Q < 0.05) break;

							// step to next sample point along light ray
							lightPoint += lightDir * V.step;

							// get next voxel index
							lightVoxelIndex = getVoxelIndex(lightPoint, V);
						}
						// accumulate color value
						newColor += (1.0f - deltaT)/k * (CF * T * Q);
						glm::clamp(newColor, 0.0f, 1.0f);
					}
				}
				// increment marching point along ray by step size
				marchPoint += V.step * glm::normalize(currentRay.direction);

				// get new voxel index for next loop
				voxelIndex = getVoxelIndex(marchPoint, V);
			}
		} 
		// blend with background color according to transmission
		glm::clamp(T, 0.0f, 1.0f);
		colors[index] = glm::mix(newColor, colors[index], T);
		colors[index] = glm::clamp(colors[index], 0.0f, 1.0f);
	}
}


// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int timestep, material* materials, int numberOfMaterials, 
					  volume* volumes, int numberOfVolumes, light* lights, int numberOfLights, Perlin* perlin1, Perlin* perlin2)
{
	// set up crucial magic
	int tileSize = 10;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	//send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
	//package volumes and send to GPU
	volume* volumeList = new volume[numberOfVolumes];
	for (int i = 0; i < numberOfVolumes; i++) {
		volume newVolume;
		newVolume.isSet			   = volumes[i].isSet;
		newVolume.volumeid         = volumes[i].volumeid;
		newVolume.materialid       = volumes[i].materialid;
		newVolume.delt             = volumes[i].delt;
		newVolume.step             = volumes[i].step;
		newVolume.xyzc             = volumes[i].xyzc;
		newVolume.translation      = volumes[i].translation;
		newVolume.rotation         = volumes[i].rotation;
		newVolume.scale            = volumes[i].scale;
		newVolume.transform        = volumes[i].transform;
		newVolume.inverseTransform = volumes[i].inverseTransform;

		voxel* cudaVolumeVoxels = NULL;
		int numVoxels = int(newVolume.xyzc.x*newVolume.xyzc.y*newVolume.xyzc.z);
		hipMalloc((void**)&cudaVolumeVoxels, numVoxels*sizeof(voxel));
		
		voxel* voxels = new voxel[numVoxels];
		for (int v = 0; v < numVoxels; v++) {
			voxels[v].density = volumes[i].voxels[v].density;
			//if (v % 1000 == 0) printf("iteration %d, voxel %d: %f\n", timestep, v, voxels[v].density);
		}
		hipMemcpy(cudaVolumeVoxels, voxels, numVoxels*sizeof(voxel), hipMemcpyHostToDevice);
		newVolume.voxels = cudaVolumeVoxels;

		volumeList[i] = newVolume;

		delete voxels;
	}
	volume* cudavolumes = NULL;
	hipMalloc((void**)&cudavolumes, numberOfVolumes*sizeof(volume));
	hipMemcpy(cudavolumes, volumeList, numberOfVolumes*sizeof(volume), hipMemcpyHostToDevice);
  

	//package materials and send to GPU
	material* materialList = new material[numberOfMaterials];
	for (int i=0; i<numberOfMaterials; i++){
		material newMaterial;
		newMaterial.color = materials[i].color;
		materialList[i] = newMaterial;
	}
	material* cudamaterials = NULL;
	hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
	hipMemcpy(cudamaterials, materialList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);
  

	// package lights and send to GPU
	light* lightList = new light[numberOfLights];
	for(int i=0; i<numberOfLights; i++){
		light newLight;
		newLight.position = lights[i].position;
		newLight.color = lights[i].color;
		lightList[i] = newLight;
	}
	light* cudalights = NULL;
	hipMalloc((void**)&cudalights, numberOfLights*sizeof(light));
	hipMemcpy(cudalights, lightList, numberOfLights*sizeof(light), hipMemcpyHostToDevice);
  

	//package perlin
	Perlin* cudaperlin1 = NULL;
	hipMalloc((void**)&cudaperlin1, sizeof(Perlin));
	hipMemcpy(cudaperlin1, perlin1, sizeof(Perlin), hipMemcpyHostToDevice);
	Perlin* cudaperlin2 = NULL;
	hipMalloc((void**)&cudaperlin2, sizeof(Perlin));
	hipMemcpy(cudaperlin2, perlin2, sizeof(Perlin), hipMemcpyHostToDevice);
	

	//package camera
	cameraData cam;
	cam.delt = renderCam->delt;
	cam.step = renderCam->step;
	cam.brgb = renderCam->brgb;
	cam.xyzc = renderCam->xyzc;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->position;
	cam.view = renderCam->view;
	cam.up = renderCam->up;
	cam.fov = renderCam->fov;


	// kernel call to populate voxel densities
	for (int i = 0; i < numberOfVolumes; i++) {
		//if (!volumes[i].isSet) {
			dim3 voxelThreadsPerBlock(tileSize, tileSize, tileSize);
			dim3 voxelFullBlocksPerGrid((int)ceil(float(volumes[i].xyzc.x)/float(tileSize)), 
										(int)ceil(float(volumes[i].xyzc.y)/float(tileSize)), 
										(int)ceil(float(volumes[i].xyzc.z)/float(tileSize)));
			voxelizeVolumeWithNoise<<<voxelFullBlocksPerGrid, voxelThreadsPerBlock>>>(i, cudavolumes, cudaperlin1, cudaperlin2, (float)timestep);
		//}
	}
	

	//kernel launches
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)timestep, cam, cudaimage, cudalights, numberOfLights, cudamaterials, 
		cudavolumes, numberOfVolumes, renderCam->iterations);
  
	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);
  

	//retrieve image from GPU
	hipMemcpy(renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
  
	// save state of volumes
	volume* volumesArr = new volume[numberOfVolumes];
	hipMemcpy(volumesArr, cudavolumes, numberOfVolumes*sizeof(volume), hipMemcpyDeviceToHost);

	for (int i = 0; i < numberOfVolumes; i++) {
		if (!volumes[i].isSet) {
			int numVoxels = int(volumes[i].xyzc.x*volumes[i].xyzc.y*volumes[i].xyzc.z);
			hipMemcpy(volumes[i].voxels, volumesArr[i].voxels, numVoxels*sizeof(voxel), hipMemcpyDeviceToHost);
			volumes[i].isSet = true;
		}
	}

	//free up stuff, or else we'll leak memory like a madman
	delete lightList;
	for (int i = 0; i < numberOfVolumes; i++) {
		hipFree( volumeList[i].voxels);
	}
	delete volumeList;
	delete volumesArr;
	delete materialList;
	hipFree( cudaimage );
	hipFree( cudalights );
	hipFree( cudavolumes );
	hipFree( cudamaterials );
	hipFree( cudaperlin1 );
	hipFree( cudaperlin2 );


	// make certain the kernel has completed
	hipDeviceSynchronize();

	checkCUDAError("Kernel failed!");
}
