#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

__host__ __device__ float generateRandomFloatFromSeed(int index, float seed){
  thrust::default_random_engine rng(hash(index*seed));
  thrust::uniform_real_distribution<float> u01(0,1);

  return (float)u01(rng);
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){

  //establish "right" camera direction
  glm::normalize(eye); glm::normalize(view);
  glm::vec3 right = glm::normalize(glm::cross(up, view));
  
  // calculate P1 and P2 in both x and y directions
  glm::vec3 image_center = eye + view;
  glm::vec3 P1_X = image_center - tan((float)4.0*fov.x)*right;
  glm::vec3 P2_X = image_center + tan((float)4.0*fov.x)*right;
  glm::vec3 P1_Y = image_center - tan((float)4.0*fov.y)*up;
  glm::vec3 P2_Y = image_center + tan((float)4.0*fov.y)*up;
  
  glm::vec3 bottom_left  = P1_X + (P1_Y - image_center);
  glm::vec3 bottom_right = P2_X + (P1_Y - image_center);
  glm::vec3 top_left     = P1_X + (P2_Y - image_center);

  glm::vec3 imgRight = bottom_right - bottom_left;
  glm::vec3 imgUp    = top_left - bottom_left;

  // supersample the pixels by taking a randomly offset ray in each iteration
  glm::vec3 random_offset = generateRandomNumberFromThread(resolution, time, x, y);
  float x_offset = random_offset.x;
  float y_offset = random_offset.y;
  glm::vec3 img_point = bottom_left + ((float)x + x_offset)/(float)resolution.x*imgRight + ((float)y + y_offset)/(float)resolution.y*imgUp;
  glm::vec3 direction = glm::normalize(img_point - eye); 

  // return value
  ray r; r.origin = eye; r.direction = direction;
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
  
	if(x<=resolution.x && y<=resolution.y){

		glm::vec3 color;
		color.x = image[index].x*255.0;
		color.y = image[index].y*255.0;
		color.z = image[index].z*255.0;

		if(color.x>255)
			color.x = 255;
		else if (color.x<0)
			color.x = 0;
		
		if(color.y>255)
			color.y = 255;
		else if (color.y<0)
			color.y = 0;
		
		if(color.z>255)
			color.z = 255;
		else if (color.z<0)
			color.z = 0;
      
		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 1.0;
		PBOpos[index].x = color.x;
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void voxelizeVolumeWithNoise(int index, volume* volumes, Perlin* perlin1, Perlin* perlin2, int timestep)
{
	// get current volume
	volume V = volumes[index];

	// get current voxel within volume
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;		// x-index
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;		// y-index
	int z = (blockIdx.z * blockDim.z) + threadIdx.z;		// z-index
	int voxelIndex = x*V.xyzc.y*V.xyzc.z + y*V.xyzc.z + z;	// overall voxel index
	voxel vox = V.voxels[voxelIndex];

	// find distance from voxel to center of volume
	glm::vec3 localPosition3D = getLocalVoxelPosition(glm::vec3((float)x, (float)y, (float)z), V);
	float length = glm::distance(localPosition3D, glm::vec3(0.0f, 0.0f, 0.0f));

	// assert voxel is within hemispheric ellipsoid
	if ( ! ((length < 0.5f) && (voxelIndex < V.xyzc.x*V.xyzc.y*V.xyzc.z) && (localPosition3D.y < 0.2)))
		return;

	// find animated noise by blending between two perlin noise functions based on frame
	float p1 = (perlin1->Get(multiplyMV(V.transform, glm::vec4(localPosition3D, 0.0))) + (1.0 - (length / 0.5f))) * (0.5f - length);
	float p2 = (perlin2->Get(multiplyMV(V.transform, glm::vec4(localPosition3D, 0.0))) + (1.0 - (length / 0.5f))) * (0.5f - length);
	float pf = max(glm::mix(p1, p2, 0.7f), 0.0f);

	// set voxel properties
	vox.density					   = pf;							// density of volume for transmittance
	vox.extinctionProbability	   = max((1.0-pf), 0.0f) * 0.9f;	// how likely is this cloud voxel to disappear
	vox.vaporProbability		   = pf * 0.003f;					// how likely is this cloud voxel to randomly accumulate vapor
	vox.phaseTransitionProbability = pf * 0.00003;					// how likely is this cloud voxel to change phase to liquid
		
	// if this is the first time, initialize simulation state
	if (!V.isSet) {
		if ((localPosition3D.y > 0.15f) && (pf > 0.01f)) { vox.states |= 0x1; }
		else											 { vox.states &= 0x0; }
	}

	// save out voxel
	V.voxels[voxelIndex] = vox;
}


//Cloud growth and dynamics (implementation adapted from Game Engine Gems 2, Chapter 2)
__global__ void updateVolume(int index, volume* volumes, float iterations)
{
	// bits for updating
	char HAS_CLOUD_BIT		  = 0x01;
	char PHASE_TRANSITION_BIT = 0x02;
	char VAPOR_BIT			  = 0x04;

	// identify current volume
	volume V = volumes[index];

	// get index of voxel within volume
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int z = (blockIdx.z * blockDim.z) + threadIdx.z;
	int voxelIndex = x*V.xyzc.y*V.xyzc.z + y*V.xyzc.z + z;
	voxel vox = V.voxels[voxelIndex];

	//----------------------------------------------------------//
	// BEGIN GAME ENGINE GEMS CODE                              //
	//----------------------------------------------------------//

	char phaseStates = 0x0;

	if (x+1 < V.xyzc.x)	{ int idx = (x+1)*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (y+1 < V.xyzc.y)	{ int idx = ( x )*V.xyzc.y*V.xyzc.z + (y+1)*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (z+1 < V.xyzc.z)	{ int idx = ( x )*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + (z+1); phaseStates |= V.voxels[idx].states; }
	if (x+2 < V.xyzc.x)	{ int idx = (x+2)*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (y+2 < V.xyzc.y)	{ int idx = ( x )*V.xyzc.y*V.xyzc.z + (y+2)*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (x-1 >= 0)		{ int idx = (x-1)*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (y-1 >= 0)		{ int idx = ( x )*V.xyzc.y*V.xyzc.z + (y-1)*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (z-1 >= 0)		{ int idx = ( x )*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + (z-1); phaseStates |= V.voxels[idx].states; }
	if (x-2 >= 0)		{ int idx = (x-2)*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (y-2 >= 0)		{ int idx = ( x )*V.xyzc.y*V.xyzc.z + (y-2)*V.xyzc.z + ( z ); phaseStates |= V.voxels[idx].states; }
	if (z-2 >= 0)		{ int idx = ( x )*V.xyzc.y*V.xyzc.z + ( y )*V.xyzc.z + (z-2); phaseStates |= V.voxels[idx].states; }

	bool phaseActivation = ((phaseStates & PHASE_TRANSITION_BIT) != 0);
	bool thisPhaseActivation = ((vox.states & PHASE_TRANSITION_BIT) != 0);

	// Set whether this cell is in a phase transition state
	double rnd = generateRandomFloatFromSeed(voxelIndex,iterations);

	bool phaseTransition = ((!thisPhaseActivation) && (vox.states & VAPOR_BIT) && phaseActivation)
							|| (rnd < vox.phaseTransitionProbability);

	if (phaseTransition) { vox.states |=  PHASE_TRANSITION_BIT; }
	else				 { vox.states &= ~PHASE_TRANSITION_BIT; }

	// Set whether this cell has acquired humidity
	rnd = generateRandomFloatFromSeed(voxelIndex,iterations);

	bool vapor = ((vox.states & VAPOR_BIT) && !thisPhaseActivation) 
					|| (rnd < vox.vaporProbability);

	if (vapor) { vox.states |=  VAPOR_BIT; }
	else	   { vox.states &= ~VAPOR_BIT; }

	// Set whether this cell contains a cloud
	rnd = generateRandomFloatFromSeed(voxelIndex,iterations);

	bool hasCloud = ((vox.states & HAS_CLOUD_BIT) || thisPhaseActivation)
					&& (rnd > vox.extinctionProbability);

	if (hasCloud) { vox.states |=  HAS_CLOUD_BIT; }
	else		  { vox.states &= ~HAS_CLOUD_BIT; }

	//----------------------------------------------------------//
	// FINISH GAME ENGINE GEMS CODE                             //
	//----------------------------------------------------------//

	V.voxels[voxelIndex] = vox;
}

//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, glm::vec3* colors, light* lights, int numberOfLights,
							material* materials, volume* volumes, int numberOfVolumes, float iterations)
{
	// Find index of pixel and create empty color vector
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	// Get initial ray from camera through this position
	ray currentRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
  
	// Return values for the intersection test
	glm::vec3 intersection_point;

	// constant attenuation for transmission
	float k = 0.2f;

	// initialize background color for blending
	colors[index] = cam.brgb * (1.0f - (y / cam.resolution.y));

	// traverse all volumes in order (not perfect)
	for (int v = 0; v < numberOfVolumes; v++)
	{
		// get the current volume
		volume V = volumes[v];

		// initialize transmission of volume at this pixel to 1.0
		float T = 1.0;
	
		// create an empty color for the volume at this pixel
		glm::vec3 newColor = glm::vec3(0.0f);

		float depth = volumeIntersectionTest(V, currentRay, intersection_point);
		if (depth > 0.0)
		{
			// initial intersection point on bounding box of volume
			glm::vec3 marchPoint = intersection_point;

			// color of volumetric material 
			glm::vec3 volCol = materials[V.materialid].color;

			// index of initial intersection point in volume density grid
			int voxelIndex = getVoxelIndex(marchPoint, V);

			// recurse through the volume and perform operations
			// while still inside (i.e. point has valid voxel index)
			while (voxelIndex >= 0) {

				// density of voxel at point
				voxel vox = V.voxels[voxelIndex];
				char voxstates = vox.states;
				if ((voxstates & 0x1) != 0x1) {
					marchPoint += V.step * glm::normalize(currentRay.direction);
					voxelIndex = getVoxelIndex(marchPoint, V);
					continue;
				}
				float p = vox.density;
			
				// transmission value at point evaluated using given function
				float deltaT = exp(-k*V.step*p);
			
				// accumulate transmission along ray
				// and break if below threshold
				T *= min(deltaT, 1.0f);
				if (T < 0.1) break;

				// calculate lighting
				if (deltaT < 1.0f) {
					for (int i = 0; i < numberOfLights; i++)
					{
						// initialize transmission along
						// light ray to zero
						float Q = 1.0;

						// ith scene light
						light L = lights[i];

						// material color scaled by light intensity
						glm::vec3 CF = volCol * L.color;

						// first sampling point along light ray is
						// march point
						glm::vec3 lightPoint = marchPoint;

						// light ray
						glm::vec3 lightDir = glm::normalize(L.position - marchPoint);
				
						// get index of voxel for point along light ray
						int lightVoxelIndex = getVoxelIndex(lightPoint, V);
				
						// recurse along light ray and perform operations
						// while still inside (i.e. point has valid voxel index
						while (lightVoxelIndex >= 0) 
						{
							// density at point along light ray
							voxel lightVoxel = V.voxels[lightVoxelIndex];
							float pLight = lightVoxel.density * (lightVoxel.states & 0x1);
					
							// light transmission value at point along light ray
							float deltaQ = exp(-k*V.step*pLight);

							// accumulate opacity of point
							Q *= deltaQ;
							if (Q < 0.1) break;

							// step to next sample point along light ray
							lightPoint += lightDir * V.step;

							// get next voxel index
							lightVoxelIndex = getVoxelIndex(lightPoint, V);
						}
						// accumulate color value
						newColor += (1.0f - deltaT)/k * (CF * T * Q);
						glm::clamp(newColor, 0.0f, 1.0f);
					}
				}
				// increment marching point along ray by step size
				marchPoint += V.step * glm::normalize(currentRay.direction);

				// get new voxel index for next loop
				voxelIndex = getVoxelIndex(marchPoint, V);
			}
		} 
		// blend with background color according to transmission
		glm::clamp(T, 0.0f, 1.0f);
		newColor = (newColor - glm::vec3(0.1)) * 1.5f;
		colors[index] = glm::mix(newColor, colors[index], T);
		colors[index] = glm::clamp(colors[index], 0.0f, 1.0f);
	}
}


// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int timestep, material* materials, int numberOfMaterials, 
					  volume* volumes, int numberOfVolumes, light* lights, int numberOfLights, Perlin* perlin1, Perlin* perlin2)
{
	// set up crucial magic
	int tileSize = 10;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	//send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
	//package volumes and send to GPU
	volume* volumeList = new volume[numberOfVolumes];
	for (int i = 0; i < numberOfVolumes; i++) {
		volume newVolume;
		newVolume.isSet			   = volumes[i].isSet;
		newVolume.volumeid         = volumes[i].volumeid;
		newVolume.materialid       = volumes[i].materialid;
		newVolume.delt             = volumes[i].delt;
		newVolume.step             = volumes[i].step;
		newVolume.xyzc             = volumes[i].xyzc;
		newVolume.translation      = volumes[i].translation + (float)timestep*volumes[i].velocity*glm::vec3(1.0, 0.0, 0.0);
		newVolume.rotation         = volumes[i].rotation;
		newVolume.scale            = volumes[i].scale;
		glm::mat4 transform = utilityCore::buildTransformationMatrix(newVolume.translation, newVolume.rotation, newVolume.scale);
		newVolume.transform = utilityCore::glmMat4ToCudaMat4(transform);
		newVolume.inverseTransform = utilityCore::glmMat4ToCudaMat4(glm::inverse(transform));
		//newVolume.transform        = volumes[i].transform;
		//newVolume.inverseTransform = volumes[i].inverseTransform;

		voxel* cudaVolumeVoxels = NULL;
		int numVoxels = int(newVolume.xyzc.x*newVolume.xyzc.y*newVolume.xyzc.z);
		hipMalloc((void**)&cudaVolumeVoxels, numVoxels*sizeof(voxel));
		
		voxel* voxels = new voxel[numVoxels];
		for (int v = 0; v < numVoxels; v++) {
			voxels[v].states = volumes[i].voxels[v].states;
			voxels[v].density = volumes[i].voxels[v].density;
			voxels[v].vaporProbability = volumes[i].voxels[v].vaporProbability;
			voxels[v].extinctionProbability = volumes[i].voxels[v].extinctionProbability;
			voxels[v].phaseTransitionProbability = volumes[i].voxels[v].phaseTransitionProbability;
		}
		hipMemcpy(cudaVolumeVoxels, voxels, numVoxels*sizeof(voxel), hipMemcpyHostToDevice);
		newVolume.voxels = cudaVolumeVoxels;

		volumeList[i] = newVolume;

		delete voxels;
	}
	volume* cudavolumes = NULL;
	hipMalloc((void**)&cudavolumes, numberOfVolumes*sizeof(volume));
	hipMemcpy(cudavolumes, volumeList, numberOfVolumes*sizeof(volume), hipMemcpyHostToDevice);
  

	//package materials and send to GPU
	material* materialList = new material[numberOfMaterials];
	for (int i=0; i<numberOfMaterials; i++){
		material newMaterial;
		newMaterial.color = materials[i].color;
		materialList[i] = newMaterial;
	}
	material* cudamaterials = NULL;
	hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
	hipMemcpy(cudamaterials, materialList, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);
  

	// package lights and send to GPU
	light* lightList = new light[numberOfLights];
	for(int i=0; i<numberOfLights; i++){
		light newLight;
		newLight.position = lights[i].position;
		newLight.color = lights[i].color;
		lightList[i] = newLight;
	}
	light* cudalights = NULL;
	hipMalloc((void**)&cudalights, numberOfLights*sizeof(light));
	hipMemcpy(cudalights, lightList, numberOfLights*sizeof(light), hipMemcpyHostToDevice);
  

	//package perlin
	Perlin* cudaperlin1 = NULL;
	hipMalloc((void**)&cudaperlin1, sizeof(Perlin));
	hipMemcpy(cudaperlin1, perlin1, sizeof(Perlin), hipMemcpyHostToDevice);
	Perlin* cudaperlin2 = NULL;
	hipMalloc((void**)&cudaperlin2, sizeof(Perlin));
	hipMemcpy(cudaperlin2, perlin2, sizeof(Perlin), hipMemcpyHostToDevice);
	

	//package camera
	cameraData cam;
	cam.delt = renderCam->delt;
	cam.step = renderCam->step;
	cam.brgb = renderCam->brgb;
	cam.xyzc = renderCam->xyzc;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->position;
	cam.view = renderCam->view;
	cam.up = renderCam->up;
	cam.fov = renderCam->fov;


	// kernel call to populate voxel densities
	for (int i = 0; i < numberOfVolumes; i++) {
		int voxelTileSize = 2;
		dim3 voxelThreadsPerBlock(voxelTileSize, voxelTileSize, voxelTileSize);
		dim3 voxelFullBlocksPerGrid((int)ceil(float(volumes[i].xyzc.x)/float(voxelTileSize)), 
									(int)ceil(float(volumes[i].xyzc.y)/float(voxelTileSize)), 
									(int)ceil(float(volumes[i].xyzc.z)/float(voxelTileSize)));
		voxelizeVolumeWithNoise<<<voxelFullBlocksPerGrid, voxelThreadsPerBlock>>>(i, cudavolumes, cudaperlin1, cudaperlin2, (float)timestep);
		updateVolume<<<voxelFullBlocksPerGrid, voxelThreadsPerBlock>>>(i, cudavolumes, timestep);
	}
	

	//kernel launches
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)timestep, cam, cudaimage, cudalights, numberOfLights, cudamaterials, 
		cudavolumes, numberOfVolumes, renderCam->iterations);
  
	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);
  

	//retrieve image from GPU
	hipMemcpy(renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
  
	// save state of volumes
	volume* volumesArr = new volume[numberOfVolumes];
	hipMemcpy(volumesArr, cudavolumes, numberOfVolumes*sizeof(volume), hipMemcpyDeviceToHost);

	for (int i = 0; i < numberOfVolumes; i++) {
		int numVoxels = int(volumes[i].xyzc.x*volumes[i].xyzc.y*volumes[i].xyzc.z);
		hipMemcpy(volumes[i].voxels, volumesArr[i].voxels, numVoxels*sizeof(voxel), hipMemcpyDeviceToHost);
		volumes[i].isSet = true;
		volumes[i].translation = volumesArr[i].translation;
	}

	//free up stuff, or else we'll leak memory like a madman
	delete lightList;
	for (int i = 0; i < numberOfVolumes; i++) {
		hipFree( volumeList[i].voxels);
	}
	delete volumeList;
	delete volumesArr;
	delete materialList;
	hipFree( cudaimage );
	hipFree( cudalights );
	hipFree( cudavolumes );
	hipFree( cudamaterials );
	hipFree( cudaperlin1 );
	hipFree( cudaperlin2 );


	// make certain the kernel has completed
	hipDeviceSynchronize();

	checkCUDAError("Kernel failed!");
}
